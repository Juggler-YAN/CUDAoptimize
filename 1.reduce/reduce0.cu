#include "hip/hip_runtime.h"
#include <iostream>

#define THREAD_PER_BLOCK 256

__global__ void reduce0(float* g_idata, float* g_odata) {

	__shared__ float sdata[THREAD_PER_BLOCK];

	// 每一个线程从全局内存装载一个元素到共享内存
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();

	// 在共享内存上执行reduce计算
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// 将该块的结果写到全局内存
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

int main() {

	// 1.申请Host内存
	const int N = 1024 * 1024 * 32;
	const int BLOCK_PER_GRID = ceil(static_cast<float>(N) / THREAD_PER_BLOCK);
	float* A_h = NULL;
	float* Aout_h = NULL;
	hipHostMalloc((void**)&A_h, N * sizeof(float));
	hipHostMalloc((void**)&Aout_h, BLOCK_PER_GRID * sizeof(float));
	// 2.申请Device内存
	float* A_d = NULL;
	float* Aout_d = NULL;
	hipMalloc((void**)&A_d, N * sizeof(float));
	hipMalloc((void**)&Aout_d, BLOCK_PER_GRID * sizeof(float));
	// 3.初始化
	for (int i = 0; i < N; ++i) {
		A_h[i] = 1;
	}
	// 4.将Host中数据拷贝到Device中
	hipMemcpy(A_d, A_h, N * sizeof(float), hipMemcpyHostToDevice);
	// 5.kernel核函数
	dim3 Grid(BLOCK_PER_GRID, 1);
	dim3 Block(THREAD_PER_BLOCK, 1);
	reduce0 << <Grid, Block >> > (A_d, Aout_d);
	// 6.将Device中数据拷贝到Host中
	hipMemcpy(Aout_h, Aout_d, BLOCK_PER_GRID * sizeof(float), hipMemcpyDeviceToHost);
	// 7.后处理
	for (int i = 0; i < BLOCK_PER_GRID; ++i) {
		if (Aout_h[i] != THREAD_PER_BLOCK) {
			std::cout << "Wrong Result!!!" << std::endl;
			break;
		}
	}
	// 8.释放内存
	hipHostFree(A_h);
	hipHostFree(Aout_h);
	hipFree(A_d);
	hipFree(Aout_d);
    
	return 0;
}