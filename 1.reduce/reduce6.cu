
#include <hip/hip_runtime.h>
/*
#include <iostream>

#define THREAD_PER_BLOCK 256

template <unsigned int blockSize>
__device__ void warpReduce(volatile float* sdata, unsigned int tid) {
	if (blockSize >= 64) sdata[tid] += sdata[tid+32];
	if (blockSize >= 32) sdata[tid] += sdata[tid+16];
	if (blockSize >= 16) sdata[tid] += sdata[tid+8];
	if (blockSize >= 8) sdata[tid] += sdata[tid+4];
	if (blockSize >= 4) sdata[tid] += sdata[tid+2];
	if (blockSize >= 2) sdata[tid] += sdata[tid+1];
}

template <unsigned int blockSize, int NUM_THREAD>
__global__ void reduce6(float* g_idata, float* g_odata, unsigned int n) {

	__shared__ float sdata[blockSize];

	// 每一个线程从全局内存装载一个元素到共享内存
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockSize*NUM_THREAD) + threadIdx.x;

	sdata[tid] = 0;

	#pragma unroll
	for (int iter = 0; iter < NUM_THREAD; ++iter) {
		sdata[tid] += g_idata[i+iter*blockSize];
	}

	__syncthreads();

	// 在共享内存上执行reduce计算
	if (blockSize >= 512) {
		if (tid < 256) {
			sdata[tid] += sdata[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			sdata[tid] += sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			sdata[tid] += sdata[tid + 64];
		}
		__syncthreads();
	}
	if (tid < 32) warpReduce<blockSize>(sdata, tid);

	// 将该块的结果写到全局内存
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

int main() {

	// 1.申请Host内存
	const int N = 1024 * 1024 * 32;
	const int BLOCK_PER_GRID = 1024;
	const int NUM_BLOCK = N / BLOCK_PER_GRID;
	const int NUM_THREAD = NUM_BLOCK / THREAD_PER_BLOCK;
	float* A_h = NULL;
	float* Aout_h = NULL;
	cudaMallocHost((void**)&A_h, N * sizeof(float));
	cudaMallocHost((void**)&Aout_h, BLOCK_PER_GRID * sizeof(float));
	// 2.申请Device内存
	float* A_d = NULL;
	float* Aout_d = NULL;
	cudaMalloc((void**)&A_d, N * sizeof(float));
	cudaMalloc((void**)&Aout_d, BLOCK_PER_GRID * sizeof(float));
	// 3.初始化
	for (int i = 0; i < N; ++i) {
		A_h[i] = 1;
	}
	// 4.将Host中数据拷贝到Device中
	cudaMemcpy(A_d, A_h, N * sizeof(float), cudaMemcpyHostToDevice);
	// 5.kernel核函数
	dim3 Grid(BLOCK_PER_GRID, 1);
	dim3 Block(THREAD_PER_BLOCK, 1);
	reduce6<THREAD_PER_BLOCK, NUM_THREAD> << <Grid, Block >> > (A_d, Aout_d, N);
	// 6.将Device中数据拷贝到Host中
	cudaMemcpy(Aout_h, Aout_d, BLOCK_PER_GRID * sizeof(float), cudaMemcpyDeviceToHost);
	// 7.后处理
	for (int i = 0; i < BLOCK_PER_GRID; ++i) {
		if (Aout_h[i] != NUM_BLOCK) {
			std::cout << "Wrong Result!!!" << std::endl;
			break;
		}
	}
	// 8.释放内存
	cudaFreeHost(A_h);
	cudaFreeHost(Aout_h);
	cudaFree(A_d);
	cudaFree(Aout_d);
    
	return 0;
}
*/