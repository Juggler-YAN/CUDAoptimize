#include "hip/hip_runtime.h"
#include <iostream>

#define THREAD_PER_BLOCK 256

__global__ void reduce2(float* g_idata, float* g_odata) {

	__shared__ float sdata[THREAD_PER_BLOCK];

	// ÿһ���̴߳�ȫ���ڴ�װ��һ��Ԫ�ص������ڴ�
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();

	// �ڹ����ڴ���ִ��reduce����
	for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// ���ÿ�Ľ��д��ȫ���ڴ�
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

int main() {

	// 1.����Host�ڴ�
	const int N = 1024 * 1024 * 32;
	const int BLOCK_PER_GRID = ceil(static_cast<float>(N) / THREAD_PER_BLOCK);
	float* A_h = NULL;
	float* Aout_h = NULL;
	hipHostMalloc((void**)&A_h, N * sizeof(float));
	hipHostMalloc((void**)&Aout_h, BLOCK_PER_GRID * sizeof(float));
	// 2.����Device�ڴ�
	float* A_d = NULL;
	float* Aout_d = NULL;
	hipMalloc((void**)&A_d, N * sizeof(float));
	hipMalloc((void**)&Aout_d, BLOCK_PER_GRID * sizeof(float));
	// 3.��ʼ��
	for (int i = 0; i < N; ++i) {
		A_h[i] = 1;
	}
	// 4.��Host�����ݿ�����Device��
	hipMemcpy(A_d, A_h, N * sizeof(float), hipMemcpyHostToDevice);
	// 5.kernel�˺���
	dim3 Grid(BLOCK_PER_GRID, 1);
	dim3 Block(THREAD_PER_BLOCK, 1);
	reduce2 << <Grid, Block >> > (A_d, Aout_d);
	// 6.��Device�����ݿ�����Host��
	hipMemcpy(Aout_h, Aout_d, BLOCK_PER_GRID * sizeof(float), hipMemcpyDeviceToHost);
	// 7.����
	for (int i = 0; i < BLOCK_PER_GRID; ++i) {
		if (Aout_h[i] != THREAD_PER_BLOCK) {
			std::cout << "Wrong Result!!!" << std::endl;
			break;
		}
	}
	// 8.�ͷ��ڴ�
	hipHostFree(A_h);
	hipHostFree(Aout_h);
	hipFree(A_d);
	hipFree(Aout_d);
    
	return 0;
}