
#include <hip/hip_runtime.h>
/*
#include <iostream>

#define THREAD_PER_BLOCK 256

__device__ void warpReduce(volatile float* sdata, int tid) {
	sdata[tid] += sdata[tid+32];
	sdata[tid] += sdata[tid+16];
	sdata[tid] += sdata[tid+8];
	sdata[tid] += sdata[tid+4];
	sdata[tid] += sdata[tid+2];
	sdata[tid] += sdata[tid+1];
}

__global__ void reduce4(float* g_idata, float* g_odata) {

	__shared__ float sdata[THREAD_PER_BLOCK];

	// 每一个线程从全局内存装载一个元素到共享内存
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i]+g_idata[i+blockDim.x];
	__syncthreads();

	// 在共享内存上执行reduce计算
	for (unsigned int s = blockDim.x/2; s > 32; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid < 32) warpReduce(sdata, tid);

	// 将该块的结果写到全局内存
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

int main() {

	// 1.申请Host内存
	const int N = 1024 * 1024 * 32;
	const int BLOCK_PER_GRID = ceil(static_cast<float>(N) / (2*THREAD_PER_BLOCK));
	float* A_h = NULL;
	float* Aout_h = NULL;
	cudaMallocHost((void**)&A_h, N * sizeof(float));
	cudaMallocHost((void**)&Aout_h, BLOCK_PER_GRID * sizeof(float));
	// 2.申请Device内存
	float* A_d = NULL;
	float* Aout_d = NULL;
	cudaMalloc((void**)&A_d, N * sizeof(float));
	cudaMalloc((void**)&Aout_d, BLOCK_PER_GRID * sizeof(float));
	// 3.初始化
	for (int i = 0; i < N; ++i) {
		A_h[i] = 1;
	}
	// 4.将Host中数据拷贝到Device中
	cudaMemcpy(A_d, A_h, N * sizeof(float), cudaMemcpyHostToDevice);
	// 5.kernel核函数
	dim3 Grid(BLOCK_PER_GRID, 1);
	dim3 Block(THREAD_PER_BLOCK, 1);
	reduce4 << <Grid, Block >> > (A_d, Aout_d);
	// 6.将Device中数据拷贝到Host中
	cudaMemcpy(Aout_h, Aout_d, BLOCK_PER_GRID * sizeof(float), cudaMemcpyDeviceToHost);
	// 7.后处理
	for (int i = 0; i < BLOCK_PER_GRID; ++i) {
		if (Aout_h[i] != THREAD_PER_BLOCK*2) {
			std::cout << BLOCK_PER_GRID << "Wrong Result!!!" << std::endl;
			break;
		}
	}
	// 8.释放内存
	cudaFreeHost(A_h);
	cudaFreeHost(Aout_h);
	cudaFree(A_d);
	cudaFree(Aout_d);
    
	return 0;
}
*/