#include "hip/hip_runtime.h"
#include <iostream>

#define THREAD_PER_BLOCK 256

#define FETCH_FLOAT2(pointer) (reinterpret_cast<float2*>(&(pointer))[0])

__global__ void vec2_add(float* a, float* b, float* c) {
    int idx = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    float2 reg_a = FETCH_FLOAT2(a[idx]);
    float2 reg_b = FETCH_FLOAT2(b[idx]);
    float2 reg_c;
    reg_c.x = reg_a.x + reg_b.x;
    reg_c.y = reg_a.y + reg_b.y;
    FETCH_FLOAT2(c[idx]) = reg_c;
}

int main() {

	// 1.����Host�ڴ�
	const int N = 1024 * 1024 * 32;
	const int BLOCK_PER_GRID = N / THREAD_PER_BLOCK;
	float* A_h = NULL;
	float* B_h = NULL;
	float* C_h = NULL;
	hipHostMalloc((void**)&A_h, N * sizeof(float));
	hipHostMalloc((void**)&B_h, N * sizeof(float));
	hipHostMalloc((void**)&C_h, N * sizeof(float));
	// 2.����Device�ڴ�
	float* A_d = NULL;
	float* B_d = NULL;
	float* C_d = NULL;
	hipMalloc((void**)&A_d, N * sizeof(float));
	hipMalloc((void**)&B_d, N * sizeof(float));
	hipMalloc((void**)&C_d, N * sizeof(float));
	// 3.��ʼ��
	for (int i = 0; i < N; ++i) {
		A_h[i] = 1;
		B_h[i] = 1;
	}
	// 4.��Host�����ݿ�����Device��
	hipMemcpy(A_d, A_h, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, N * sizeof(float), hipMemcpyHostToDevice);
	// 5.kernel�˺���
	dim3 Grid(BLOCK_PER_GRID/2, 1);
	dim3 Block(THREAD_PER_BLOCK, 1);
	for (int i = 0; i < 10; ++i) {
		vec2_add << <Grid, Block >> > (A_d, B_d, C_d);
	}
	// 6.��Device�����ݿ�����Host��
	hipMemcpy(C_h, C_d, N * sizeof(float), hipMemcpyDeviceToHost);
	// 7.����
	for (int i = 0; i < N; ++i) {
		if (C_h[i] != 2) {
			std::cout << "Wrong Result!!!" << std::endl;
			break;
		}
	}
	// 8.�ͷ��ڴ�
	hipHostFree(A_h);
	hipHostFree(B_h);
	hipHostFree(C_h);
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
    
	return 0;
}